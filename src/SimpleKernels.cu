#include "hip/hip_runtime.h"
#include "Config.h"
#include "SimpleKernels.h"
#include "LaunchConfiguration.h"
namespace CuGLView
{
    __global__ void K_FillBuffer(int* p, int color, int hei, int wid)
    {
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        if (i < hei && j < wid)
        {
            *(p + i*wid+j) = color;
        }
    }
    
    void FillBuffer(int* p, int color, int hei, int wid)
    {    
        dim3 block, grid;
        GetConfiguration(&block, &grid, hei, wid);
        K_FillBuffer<<<grid, block>>>(p, color, hei, wid);
    }
}